#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <assert.h>

#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "vptree.h"

int *idx_host;
__device__ tree_array *vp_tree;

vptree* newnode(double *point ,int index ,double median , int d)
{
	vptree *node = (vptree*) malloc(sizeof(vptree));
	node->idx = index;
	node->vantage_point = (double *) malloc(sizeof(double)*d);
	for (int i = 0; i < d; i++)
	{
		node->vantage_point[i] = point[index*d+i];
	}
	node->median_value = median;
	node->outer = node->inner = NULL ;
	return node;
}

vptree * getInner(vptree * T)
{
	return T->inner;
}

vptree * getOuter(vptree * T)
{
	return T->outer;
}

double getMD(vptree * T)
{
	return T->median_value;
}

double * getVP(vptree * T)
{
	return T->vantage_point;
}

int getIDX(vptree * T)
{
	return T->idx;
}

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

int* set_start_end(int* milestone,int level);
int* set_sorting_priority(int *milestone,int itterations,int n);
__global__ void block_calculate_distances(double *shared_points_cuda,double* distances_cuda, int *idx_cuda , int* cuda_milestone,int d ,int power_of_level);
__global__ void set_median(double* distances_cuda,int* cuda_milestone,int power_of_level);
__device__ void calculate_distance_gpu(double *shared_points_cuda,double* distances_cuda,int *idx_cuda ,double *vantage_point,int start , int  end , int d );
//__global__ void update(double *distances_cuda,int *cuda_milestone,int *temp,int power_of_level);
vptree *build_tree(tree_array *host,vptree *node ,double *data,int index,int d, int n);

vptree *buildvp(double *X,int n, int d)
{
	struct timeval startwtime,endwtime;
	double seq_time;

	tree_array *dynamic_vp_tree;
	int itterations = ((int)log2f(n))+1;
	int num = pow(2,itterations);
	checkCuda( hipMalloc( (void **) &dynamic_vp_tree,sizeof(tree_array)* num));

	gettimeofday(&startwtime,NULL);

	buildvp_cuda(X, dynamic_vp_tree ,n, d);

	gettimeofday(&endwtime,NULL);
	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
	printf("GPU TREE CONSTRUCTION:%lf \n",seq_time );
	
	tree_array *host_tree = (tree_array*) malloc(sizeof(tree_array)*num);
	checkCuda(hipMemcpy(host_tree,dynamic_vp_tree,sizeof(tree_array)*num,hipMemcpyDeviceToHost));
	vptree *root;
	root = build_tree(host_tree,root ,X,0,d,num);
 
	free(host_tree);
	hipFree(dynamic_vp_tree);
	return root;
}

/*The conctruction of the vantage point tree in the gpu. It stores it in tree_array stucture.
It constructs the tree level by level, calculating the parameters of each node in parallel.*/
void buildvp_cuda(double *X, tree_array *cuda_tree ,int n, int d)
{	
	struct timeval startwtime,endwtime;
	double seq_time;
	
 	int itterations = ((int)log2f(n))+1;
 	int num = pow(2,itterations);
 	printf("\n");
 	printf("N=%d and Itterations:%d \n", n , itterations );

	checkCuda(hipHostMalloc((void **) &idx_host , n*sizeof(int)));
  

	/*********** Data initialization*******************/
   	printf("\n");

	gettimeofday(&startwtime,NULL);
	//Seting IDs
	for (int i=0;i<n;i++){
    	idx_host[i]=i;
	}
	
	//Cuda array to store the points
	double *dynamic_shared_points_cuda;
	checkCuda(hipMalloc((void **) &dynamic_shared_points_cuda , n*d*sizeof(double)));
    checkCuda(hipMemcpy(dynamic_shared_points_cuda,X,sizeof(double)*n*d,hipMemcpyHostToDevice));
	
	//Cuda array to store the distances
	int *dynamic_idx_cuda;
	checkCuda(hipMalloc((void **) &dynamic_idx_cuda , n*sizeof(int)));
    checkCuda(hipMemcpy(dynamic_idx_cuda,idx_host,sizeof(int)*n,hipMemcpyHostToDevice));

    //Cuda array to store the distances
	double *dynamic_dist_cuda;
	checkCuda(hipMalloc((void **) &dynamic_dist_cuda,sizeof(double)*n));

	//Connect the pointer with the global variable
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(vp_tree), &cuda_tree, sizeof(tree_array*) ,0,hipMemcpyHostToDevice));
	
	//Milestone array stores the start and the end index of each segment in which we cut our points in every level
	int milestone_size = (int) 4*pow(2,(itterations));
	int *milestone = (int *)malloc(sizeof(int)*milestone_size);
	*milestone=0;
	*(milestone+1)=n; 
	
	//Cuda variable for milestone
	int *cuda_milestone;
	checkCuda(hipMalloc((void **) &cuda_milestone , milestone_size*sizeof(int)));
	
	int *temp_milestone;
	checkCuda(hipMalloc((void **) &temp_milestone , milestone_size*sizeof(int)));

	//Group array stores the group of each point.With this way we can sort each group seperatly.
	//I got this idea based on the following link.
	//https://stackoverflow.com/questions/28150098/how-to-use-thrust-to-sort-the-rows-of-a-matrix?fbclid=IwAR2wAiz9aaGKAMZ5gnAuJKa81dP7qn-CsnA-w932911qj2cBClTh-88CND8
	int *group = (int*) malloc(sizeof(int)*n);
	
	hipDeviceSetLimit(hipLimitMallocHeapSize, (1024*1024*1024));

	gettimeofday(&endwtime,NULL);
	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
   	printf("Setting up variables = %f \n", seq_time);
	
	double distances_time =0.0;  	//time to measure the total distances calculation
	double median_time =0.0; 	 	//time to measure the total median selection
	printf("\n");
	printf("Starting....\n");
	printf("\n");
	//Main procedure: Build the tree level by level. In each level calculate the parameters of each node in parallel.
	for (int level = 0; level < itterations; level++)
	{
		int it_num = (int) pow(2,level); 	//max nodes in each level
		
		//Get time to measure distances calculation
		gettimeofday(&startwtime,NULL);
		
		//Calculating the start and the end of each segment based on the previous ones
		milestone = set_start_end(milestone,level);
		checkCuda(hipMemcpy(cuda_milestone,milestone,sizeof(int)*milestone_size,hipMemcpyHostToDevice));
		//checkCuda(hipMemcpy(milestone,cuda_milestone,sizeof(int)*milestone_size,hipMemcpyDeviceToHost));

		/*Caclculate distances of every node in parallel*/
		block_calculate_distances<<<512,1024,sizeof(double)*d>>>( dynamic_shared_points_cuda,dynamic_dist_cuda, dynamic_idx_cuda ,cuda_milestone,d , it_num);
		checkCuda(hipGetLastError());
		checkCuda(hipDeviceSynchronize());

		//Measure the time elapsed and accumulate it in the total time
		gettimeofday(&endwtime,NULL);
		seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
		distances_time +=seq_time;

		//Get time to measure medians selection
		gettimeofday(&startwtime,NULL);

		//Seperate each group of points
		group = set_sorting_priority(milestone,pow(2,level),n);

		/********Vectors initiation***********/
		thrust::host_vector<int> h_segments(group,group+n);		//Host groups
  		thrust::device_vector<int> d_segments = h_segments;		//Device groups

  		thrust::device_vector<double> d_result(dynamic_dist_cuda,dynamic_dist_cuda+n);		//Device vector of distances
  		thrust::device_vector<int> d_result_idx(dynamic_idx_cuda,dynamic_idx_cuda+n);		//Device vector of IDs

  		//Sorting distances and update the tuple = {groups , ids} and the resort the groups and update the tuple = {distances , ids}
  		thrust::stable_sort_by_key(d_result.begin(), d_result.end(), thrust::make_zip_iterator(thrust::make_tuple(d_segments.begin(),d_result_idx.begin())) );
 	 	hipDeviceSynchronize();
 	 	thrust::stable_sort_by_key(d_segments.begin(), d_segments.end(), thrust::make_zip_iterator(thrust::make_tuple(d_result.begin(),d_result_idx.begin()))  );
  		hipDeviceSynchronize();

  		//Save back the distances and the ids
  		thrust::copy(d_result_idx.begin(),d_result_idx.end(),dynamic_idx_cuda);
  		thrust::copy(d_result.begin(),d_result.end(),dynamic_dist_cuda);
		hipDeviceSynchronize();
		
		//Setting the median value of each node in parallel.
		set_median<<<1,1024>>>(dynamic_dist_cuda,cuda_milestone,it_num);
		hipDeviceSynchronize();


		gettimeofday(&endwtime,NULL);
		seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
		median_time +=seq_time;
	}

	printf("Total distance calculation time: %lf \n", distances_time);
	printf("Picking(sorting) and setting median: %lf \n", median_time);

	free(milestone);
	hipFree(dynamic_dist_cuda);
	hipFree(dynamic_idx_cuda);
	hipFree(dynamic_shared_points_cuda);
	hipFree(cuda_milestone);
}

//Move the true from device to host and transform it to vpree struct
vptree *build_tree(tree_array *host,vptree *node ,double *data,int index,int d, int n)
{
	if (index>n-1)
		return NULL;
	else{
		if (host[index].valid==false)
			return NULL;
		else{
			if (index%2==0)
			{
				if (host[index].idx==host[index-1].idx || host[index].idx==host[(index-2)/2].idx)
				{
					host[index].valid=false;
					return NULL;
				}
			}
			else
			{
				if (host[index].idx==host[(index-1)/2].idx)
				{
					host[index].valid=false;
					return NULL;
				}
			}
			node = newnode(data,host[index].idx, host[index].median , d);
			node->outer = build_tree(host, node->outer, data,2*index+2,d,  n);
			node->inner = build_tree(host  ,node->inner , data,2*index+1,d ,   n);
			return node;
		}
	}
}

//Find the start and the end of each segment
int* set_start_end(int* milestone,int level)
{
	int h = pow(2,level);
	int *temp=(int*)malloc(sizeof(int)*2*h);
	int start,end,n;
	if(level==0)
	{
		temp[0]=0;
		temp[1]=milestone[1]-1;
	}
	else
	{
		for (int i = 0; i < pow(2,level-1); i++)
		{
			start = milestone[2*i];
			end = milestone[2*i+1];
			n = end-start+1;
			//printf("start: %d end: %d n: %d\n", start , end , n );
		
			if(end-start==0)
			{
				temp[4*i] = start;
				temp[4*i+1] = end;
				temp[4*i+2] = start;
				temp[4*i+3] = end;
			}
			//In this occasion we have two points in the array. One of them will
			//be chosen as a vantage point,so it is going to be created a complete binary
			//which means there are gonna be nodes with only a right child node
			else if(end-start==1)
			{
				temp[4*i] = start;
				temp[4*i+1] = end-1;
				temp[4*i+2] = start;
				temp[4*i+3] = end-1;	
			}
			//In this a occasion we are having three points and the tree gonna be splitted in a perfect binary tree 
			else if(end-start==2)
			{
				temp[4*i] = start;
				temp[4*i+1] = end-2;
				temp[4*i+2] = start+1;
				temp[4*i+3] = end-1;
			}
			//General occasion
			else
			{
				temp[4*i] = start;
				if ((n-1)%2==0)
				{
					temp[4*i+1] = start+(n-1)/2-1;
					temp[4*i+2] = start+(n-1)/2;
				}
				else
				{
					temp[4*i+1] = start+(n-1)/2;
					temp[4*i+2] = start+(n-1)/2+1;
				}
				temp[4*i+3] = end-1;
 			}	
		}
	}

	for (int j = 0; j < 2*pow(2,level); j++)
	{
		milestone[j]=temp[j];
	}
	free(temp);
	return milestone;
}

//Set an ascending number to each diffirent segment
int* set_sorting_priority(int *milestone,int itterations,int n)
{
	int flag=0;
	int *array = (int*)malloc(sizeof(int)*n);
	for (int i = 0; i < itterations; i++)
	{
		int start = milestone[2*i];
		int end = milestone[2*i+1];
		if (i==0)
		{
			for (int j = start; j <= end; j++)
			{
				array[j] = flag;
				if (start==end)
				{
					flag++;
				}
				else if(j==end)
				{
					flag++;
					array[j]=flag;
					flag++;
				}
			}
		}
		else
		{
			int milestone_end =  milestone[2*i-1];
			if (start-milestone_end>1)
			{
				for (int j = milestone_end; j < start; j++)
				{
					array[j] = flag;
					flag++;
				}
			}
			for (int j = start; j <= end; j++)
			{
				array[j] = flag;
				if (start==end)
				{
					flag++;
				}
				else if(j==end)
				{
					flag++;
					array[j]=flag;
					flag++;
				}
			}
		}
		if(i==itterations-1)
		{
			for (int j = end; j < n; j++)
			{
				array[j]=flag;
				flag++;	
			}
		}
	}	
	return array;
}

//Function that set the parameters and calculate the distances for each node in parallel.
__global__ void block_calculate_distances(double* shared_points_cuda,double* distances_cuda, int *idx_cuda ,int* cuda_milestone,int d , int power_of_level)
{
	__shared__ int start;
	__shared__ int end;
	__shared__ int index;
	__shared__ int dim;
	//extern __shared__ double vp[];
	
	for (int i = 0; i < power_of_level/gridDim.x+1; ++i)
	{	
		int id = blockIdx.x+i*gridDim.x;
		if (id<power_of_level)
		{
			if (threadIdx.x==0)
			{
				dim = d;
				start =  cuda_milestone[2*id];
				end = cuda_milestone[2*id+1];
				index=power_of_level-1+id;
				/*for (int i = 0; i < dim; i++)
				{
					vp[i] = shared_points_cuda[idx_cuda[end]*dim+i];
				}*/
				vp_tree[index].idx = idx_cuda[end];
				vp_tree[index].array_idx=index;
				vp_tree[index].valid=true;
				if(index>0){
					if (index%2==0){
						if (vp_tree[index].idx==vp_tree[index-1].idx || vp_tree[index].idx==vp_tree[(index-2)/2].idx)
							vp_tree[index].valid=false;
					}
					else{
						if (vp_tree[index].idx==vp_tree[(index-1)/2].idx)
							vp_tree[index].valid=false;
					}
				}
			}
			__syncthreads();	

			calculate_distance_gpu(shared_points_cuda,distances_cuda,idx_cuda,shared_points_cuda+idx_cuda[end]*dim,start,end,dim);
			//__syncthreads();
		}
	}
}

//Device Function to calculate the distances
__device__ void calculate_distance_gpu(double* shared_points_cuda,double* distances_cuda, int *idx_cuda ,double *vantage_point,int start , int  end , int d )
{
	int n=end-start+1;
	for(int i = 0; i < n/blockDim.x+1;i++){
		int index = (start + threadIdx.x) + i*blockDim.x;
		double sum = 0;
		if (index>=start && index<=end)
		{
			for(int j=0 ; j<d ;j++){
				sum+= pow(shared_points_cuda[idx_cuda[index]*d+j] - vantage_point[j],2);
			}
			distances_cuda[index] = sqrt(sum);
		}
	}
}

//Setting the median value to each node in parallel.
__global__ void set_median(double* distances_cuda,int* cuda_milestone , int power_of_level)
{
	for (int i = 0; i < power_of_level/blockDim.x+1; i++)
	{
		int idx;
		idx=threadIdx.x+i*blockDim.x;
		if (idx<power_of_level){
			
			int index;
			index=power_of_level-1+idx;
			
			int start,end;
			start =  cuda_milestone[2*idx];
			end = cuda_milestone[2*idx+1];

			int median_index;
			if (end-start>0)
			{
				if(end-start>1)
				{
					if (end-start==2)
						median_index = start;
					else if((end-start)%2==0)
						median_index = start+(end-start)/2-1;
					else if((end-start)%2!=0)
						median_index = start+(end-start)/2;
				}	
				else if ((end-start)==1)
					median_index=start;

				vp_tree[index].median = distances_cuda[median_index];
			}
		}
	}
	__syncthreads();
}

/*Function to fix the problem of the same points.Unfortunatly it can't fix the problem of the same distance of two diffirent points.That's why it's not included*/
/*__global__ void update(double *distances_cuda,int *cuda_milestone,int *temp,int power_of_level)
{

	for (int i = 0; i < power_of_level/blockDim.x+1; i++)
	{
		int index = threadIdx.x+i*blockDim.x;
		if (index<power_of_level)
		{
			int start,end;
			start =  cuda_milestone[2*i];
			end = cuda_milestone[2*i+1];

			int median_index;
			if (end-start>0)
			{
			if(end-start>1)
				{
					if (end-start==2)
						median_index = start;
					else if((end-start)%2==0)
						median_index = start+(end-start)/2-1;
					else if((end-start)%2!=0)
						median_index = start+(end-start)/2;
				}	
				else if ((end-start)==1)
					median_index=start;
			}

			int count=median_index;

			if(end-start==0)
			{
				temp[4*index+0] = start;
				temp[4*index+1] = end;
				temp[4*index+2] = start;
				temp[4*index+3] = end;
			}
			else if(end-start==1)
			{
				temp[4*index+0] = start;
				temp[4*index+1] = end-1;
				temp[4*index+2] = start;
				temp[4*index+3] = end-1;	
			}
			else{
				while(distances_cuda[median_index]==distances_cuda[count+1])
				{
					count++;
				}
				if(end-start==2)
				{
					if (count==median_index )
					{
						temp[4*index+0] = start;
						temp[4*index+1] = end-2;
						temp[4*index+2] = start+1;
						temp[4*index+3] = end-1;
					}
					else
					{
						temp[4*index+0] = start;
						temp[4*index+1] = count;
						temp[4*index+2] = end;
						temp[4*index+3] = end;
					}
				}
				else{	
					temp[4*index+0] = start;
					temp[4*index+1] = count;
					temp[4*index+2] = count +1;
					temp[4*index+3] = end -1;
				}
			}
		}
	}

	__syncthreads();

	for (int i = 0; i < power_of_level/(blockDim.x)+1; i++)
	{
		int index = threadIdx.x + i*blockDim.x;
		if (index<power_of_level)
		{
			cuda_milestone[4*index]=temp[4*index+0];
			cuda_milestone[4*index+1]=temp[4*index+1];
			cuda_milestone[4*index+2]=temp[4*index+2];
			cuda_milestone[4*index+3]=temp[4*index+3];
		}	
	}
}*/

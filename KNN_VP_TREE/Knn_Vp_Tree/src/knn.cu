#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include <time.h>
#include <sys/time.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <algorithm>
#include "knn.h"
#include "vptree.h"
#include "cuda_stack.h"

using namespace std;

inline hipError_t cudacheck(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
/* Function to sort an array using insertion sort*/
__device__ void insertionSort(double *arr, int *idx ,int n) 
{ 
    int i, idx_key, j;
    double key; 
    for (i = 1; i < n; i++) { 
        key = arr[i];
        idx_key = idx[i]; 
        j = i - 1; 
  
        /* Move elements of arr[0..i-1], that are 
          greater than key, to one position ahead 
          of their current position */
        while (j >= 0 && arr[j] > key) { 
            arr[j + 1] = arr[j];
            idx [j + 1] = idx[j]; 
            j = j - 1; 
        } 
        arr[j + 1] = key;
        idx[j+1] = idx_key; 
    } 
}
__global__ void vp_tree_search(double *cuda_share_points,double* cuda_distances, int *cuda_idx, tree_array *cuda_tree,int index ,int n , int m , int k , int d);
__device__ void update(double *cuda_distances,int *cuda_idx ,double dist , int id , double *radius  ,int k ,int block_idx);
__device__ double calculate_dist(double *a, double *b , int d);

knnresult kNN(double* X,double* Y,int n,int m,int d,int k)
{
	struct timeval startwtime,endwtime;
	double seq_time;

	knnresult result;
	result = init_knnresult(m,k);

	double *d_shared_points;
	cudacheck(hipMalloc((void **) &d_shared_points , n*d*sizeof(double)));
    cudacheck(hipMemcpy(d_shared_points,X,sizeof(double)*n*d,hipMemcpyHostToDevice));

	int *d_idx;
	cudacheck(hipMalloc((void **) &d_idx , m*k*sizeof(int)));

	double *d_dist;
	cudacheck(hipMalloc((void **) &d_dist,sizeof(double)*k*m));
	cudacheck(hipMemcpy(d_dist,result.ndist,sizeof(double)*k*m,hipMemcpyHostToDevice));

	tree_array *dynamic_vp_tree;
	int itterations = ((int)log2f(n))+1;
	int num = pow(2,itterations);
	cudacheck( hipMalloc( (void **) &dynamic_vp_tree,sizeof(tree_array)*num));

	printf("Building vantage point tree...\n");
	gettimeofday(&startwtime,NULL);

	buildvp_cuda(X, dynamic_vp_tree ,n, d);
	
	cudacheck(hipPeekAtLastError());
	cudacheck(hipDeviceSynchronize());
	
	gettimeofday(&endwtime,NULL);
	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
   	printf("Building tree time = %f \n", seq_time);
	


   	printf("Start searching the vantage point tree\n");

	gettimeofday(&startwtime,NULL);

	cudacheck(hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024));

	vp_tree_search<<<512,1>>>(d_shared_points,d_dist,d_idx, dynamic_vp_tree , 0 , num-1 , m , k , d);
	cudacheck(hipPeekAtLastError());
	cudacheck(hipDeviceSynchronize());

	gettimeofday(&endwtime,NULL);
	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
   	printf("GPU Knn time = %f \n", seq_time);
	
	cudacheck(hipMemcpy(result.ndist, d_dist ,sizeof(double)*k*m,hipMemcpyDeviceToHost));
	cudacheck(hipMemcpy(result.nidx,d_idx,sizeof(int)*k*m,hipMemcpyDeviceToHost));

	return result;
}

knnresult init_knnresult(int m_arg , int k_arg )
{
	knnresult result;
	result.nidx = (int *) malloc(sizeof(int)*m_arg*k_arg);
	result.ndist = (double *) malloc(sizeof(double)*m_arg*k_arg);
	for (int i = 0; i < m_arg*k_arg; i++)
	{
		result.ndist[i] = INFINITY;
	}
	result.m =m_arg;
	result.k=k_arg;
	return result;
}

__global__ void vp_tree_search(double *cuda_share_points,double* cuda_distances, int *cuda_idx, tree_array *cuda_tree,int index ,int n , int m , int k , int d)
{
	for (int i = 0; i < m/gridDim.x+1; i++)
	{
		int m_id = blockIdx.x+i*gridDim.x;
		if (m_id<m)
		{
			int tree_index = index;
			//Maybe it will needed more than 1000 stack size for large dataset.
			Stack* stack = createStack(1000);
			//printf("stack created in every block\n"); 
			push(stack,cuda_tree+index);
	
			tree_array *node;
			node = (tree_array*)malloc(sizeof(tree_array));

			double *radius = (double*)malloc(sizeof(double));
			*radius = INFINITY;

			while(!isEmpty(stack))
			{
				if(!pop(stack,node))
					printf("Error:Stack is is empty\n");
				if(node->valid){
					double dist;
					dist = calculate_dist(cuda_share_points+m_id*d,cuda_share_points+(node->idx)*d,d);
					//store point and distance
					if (dist < *radius)
						update(cuda_distances, cuda_idx ,dist, node->idx,radius,k,m_id);
					//check inside
					if (dist <= node->median + *radius){
						tree_index = 2*node->array_idx + 1;
						if(tree_index<n)
							push(stack ,cuda_tree+tree_index );			
					}
					//check outside
					if (dist > node->median - *radius){
						tree_index = (2*node->array_idx + 2); 
						if(tree_index<n)
							push(stack , cuda_tree+tree_index);
					}
				}
			}
			free(node);
			free(radius);
			deleteStack(stack);
		}
	}
}

__device__ void update(double *cuda_distances,int *cuda_idx ,double dist , int id , double *radius  ,int k ,int block_idx)
{
	double temp_rad;	
	cuda_distances[block_idx*k+(k-1)] = dist;
	cuda_idx[block_idx*k+(k-1)] = id;
	insertionSort(cuda_distances+block_idx*k,cuda_idx+block_idx*k,k);
	temp_rad = cuda_distances[block_idx*k+(k-1)];
	
	*radius = temp_rad;
}

__device__ double calculate_dist(double *a, double *b , int d)
{
	double dist;
	double sum = 0;
	for (int i = 0; i < d; i++)
	{
		sum+= pow(a[i]-b[i],2);
	}
	dist = sqrt(sum);
	return dist;
}
#include "hip/hip_runtime.h"
/*!
  \file   tester.c
  \brief  Validate kNN ring implementation.

  \author Dimitris Floros
  \date   2019-11-13
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include "knn.h"
#include "vptree.h"
#include "tester_helper.h"

using namespace std;

int main(int argc, char** argv)
{
  if(argc!=4){
    printf("\nNeed 3 argument. N:number of elements D:dimensions of elements K:number of nearest neighbors");
    exit(0);
  }

  int n=atoi(argv[1]);//data
  int d=atoi(argv[2]);//dimensions
  int k=atoi(argv[3]);//number of dimensions

  struct timeval startwtime, endwtime;
  double seq_time;

  double  * corpus = (double * ) malloc( n*d * sizeof(double) );

  for (int i=0;i<n*d;i++)
    corpus[i] = ( (double) (rand()) ) / 100000000;

  gettimeofday (&startwtime, NULL);
  knnresult knnres; 
  knnres = kNN(corpus, corpus, n, n, d, k );
  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
  printf("Overall clock time = %f %d elements \n", seq_time , n);

  int isValidC = validateResult( knnres, corpus, corpus, n, n, d, k, COLMAJOR );

  int isValidR = validateResult( knnres, corpus, corpus, n, n, d, k, ROWMAJOR );
  
  printf("Tester validation: %s NEIGHBORS\n",
         STR_CORRECT_WRONG[isValidC||isValidR]);

  free( corpus );
  return 0;
  
}
